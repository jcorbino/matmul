// nvcc --compiler-options -Wall matmul.cu -o matmul
// ./matmul <ROWS_A> <COLS_B> <COLS_A>


#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>

#define TILE_WIDTH 16 // 16x16 = 256 CUDA threads per block

using namespace std;

// Macro to check for errors
#define checkCudaErrors(value)    \
    {                             \
        check((value), __LINE__); \
    }
inline void check(hipError_t code, int line)
{
    if (code != hipSuccess)
    {
        cerr << hipGetErrorString(code) << ", at line: " << line << endl;
        exit(code);
    }
}

// Kernel
__global__ void matmul(const double *A, const double *B, double *C, uint hA, uint wA, uint wB)
{
    __shared__ double As[TILE_WIDTH][TILE_WIDTH];
    __shared__ double Bs[TILE_WIDTH][TILE_WIDTH];

    uint tx = threadIdx.x;
    uint ty = threadIdx.y;
    uint bx = blockIdx.x;
    uint by = blockIdx.y;

    uint row = by * TILE_WIDTH + ty;
    uint col = bx * TILE_WIDTH + tx;

    double value = 0.0;

    for (uint ph = 0; ph < (wA + TILE_WIDTH - 1) / TILE_WIDTH; ++ph)
    {
        if (row < hA && ph * TILE_WIDTH + tx < wA)
            As[ty][tx] = A[row * wA + ph * TILE_WIDTH + tx];
        else
            As[ty][tx] = 0.0;

        if (col < wB && ph * TILE_WIDTH + ty < wA)
            Bs[ty][tx] = B[(ph * TILE_WIDTH + ty) * wB + col];
        else
            Bs[ty][tx] = 0.0;

        __syncthreads();

        for (uint k = 0; k < TILE_WIDTH; ++k)
            value += As[ty][k] * Bs[k][tx];

        __syncthreads();
    }

    if (row < hA && col < wB)
        C[row * wB + col] = value;
}

int main(int argc, char *argv[])
{
    // Host and device pointers
    double *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    uint mem_size_A, mem_size_B, mem_size_C;

    if (argc != 4)
    {
        cerr << "usage: ./matmul <ROWS_A> <COLS_B> <COLS_A>\n";
        return 1;
    }

    // Random numbers
    mt19937_64 rnd(random_device{}());
    uniform_real_distribution<double> dist(0, 1);

    // Dimensions
    uint m = atoi(argv[1]); // Rows of A, C
    uint n = atoi(argv[2]); // Cols of B, C
    uint o = atoi(argv[3]); // Cols of A, Rows of B

    mem_size_A = sizeof(double) * m * o;
    mem_size_B = sizeof(double) * o * n;
    mem_size_C = sizeof(double) * m * n;

    // Allocate memory on the host
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A, hipHostMallocDefault));
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B, hipHostMallocDefault));
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C, hipHostMallocDefault));

    // Allocate memory on the device (GPU)
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // Fill matrix A with random numbers
    for (uint k = 0; k < m * o; ++k)
        h_A[k] = dist(rnd);

    // Fill matrix B with random numbers
    for (uint k = 0; k < o * n; ++k)
        h_B[k] = dist(rnd);

    auto start = chrono::steady_clock::now();

    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(TILE_WIDTH, TILE_WIDTH);
    dim3 grid((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);

    matmul<<<grid, threads>>>(d_A, d_B, d_C, m, o, n);
    checkCudaErrors(hipPeekAtLastError());

    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    auto stop = chrono::steady_clock::now();
    cout << "Elapsed time (including data transfers): " << chrono::duration_cast<chrono::milliseconds>(stop - start).count() << " ms\n";

// Check result!
#ifdef DEBUG
    double *tmp = (double *)calloc(m * n, sizeof(double));
    for (uint i = 0; i < m; ++i)
        for (uint k = 0; k < o; ++k)
            for (uint j = 0; j < n; ++j)
                tmp[i * n + j] += h_A[i * o + k] * h_B[k * n + j];

    for (uint i = 0; i < m; ++i)
        for (uint j = 0; j < n; ++j)
            if (fabs(h_C[i * n + j] - tmp[i * n + j]) > 0.01)
                cout << "Matrices differ!\n";

    free(tmp);
#endif

    // Clean up memory
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    return 0;
}
